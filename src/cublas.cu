#include "../include/cublas.hpp"

void cublas_manage_error(int error_number)
{
	if(error_number == 1)
	{
		printf("Bad transformation type");
	}
}

hipblasHandle_t init_cublas()
{

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	return handle;

}

hipblasOperation_t convert_to_cublas_trans(char trans)
{
	hipblasOperation_t ret;
	if(trans == 'N' || trans == 'n'){
		 ret = HIPBLAS_OP_N; 
	}else if(trans == 'C' || trans == 'c')
	{
		ret = HIPBLAS_OP_C;
	}else if(trans == 'T' || trans == 't')
	{
		ret = HIPBLAS_OP_T;
	}else
	{
		cublas_manage_error(1);
	}	
	
	return ret;
}



hipblasDiagType_t convert_diag(char* DIAG)
{
	hipblasDiagType_t ret;
	if(DIAG == 'N' || DIAG == 'n')
		ret = HIPBLAS_DIAG_NON_UNIT;
	else if(DIAG == 'U' || DIAG == 'u')
		ret = HIPBLAS_DIAG_UNIT;

	return ret;
}

hipblasSideMode_t convert_side(char* SIDE)
{
	hipblasSideMode_t ret;
	if(SIDE == 'L' || side == 'l')
		ret = HIPBLAS_SIDE_LEFT;
	else if(SIDE == 'r' || SIDE == 'R')
		ret = HIPBLAS_SIDE_RIGHT;
	return ret
}
hipblasFillMode_t convert_fill(char* UPLO)
{
	hipblasFillMode_t ret;
	if(UPLO == 'U' || UPLO == 'u')
		ret == HIPBLAS_FILL_MODE_UPPER;
	else if(UPLO == 'L' || UPLO == 'l')
		ret == HIPBLAS_FILL_MODE_LOWER;
	return ret;

}




//XGEMV


void sgemv(char transA, int m, int n, float alpha, float* A, int lda, float* X, int incx, float beta, float* Y, int incy); 
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasSgemv(handle, convert_to_cublas_trans(transA), m, n, &alpha, A, lda, X, incx, &beta, Y, incy);


}
void dgemv(char transA, int m, int n, double alpha, double* A, int lda, double* X, int incx, double beta, double* Y, int incy) ;
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasDgemv(handle, convert_to_cublas_trans(transA), m, n, &alpha, A, lda, X, incx, &beta, Y, incy);


}

void cgemv(char transA, int m, int n, float_complex alpha, float_complex* A, int lda, float_complex* X, int incx, float_complex beta, float_complex* Y, int incy);
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasCgemv(handle, convert_to_cublas_trans(transA), m, n, &alpha, A, lda, X, incx, &beta, Y, incy);


}

void zgemv(char transA, int m, int n, double_complex alpha, double_complex* A, int lda, double_complex* X, int incx, double_complex beta, double_complex* Y, int incy);
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasZgemv(handle, convert_to_cublas_trans(transA), m, n, &alpha, A, lda, X, incx, &beta, Y, incy);


}



//XGBMV

void sgbmv(char transA, int m, int n, int kl, int ku, float alpha, float* A, int lda, float* X, int incx, float beta, float* Y, int incy)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasSgbmv(handle, convert_to_cublas_trans(transA), m, n, kl, ku, &alpha, A, lda, X, incx, &beta, Y, incy);


}

void dgbmv(char transA, int m, int n, int kl, int ku, double alpha, double* A, int lda, double* X, int incx, double beta, double* Y, int incy) 
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasDgbmv(handle, convert_to_cublas_trans(transA), m, n, kl, ku, &alpha, A, lda, X, incx, &beta, Y, incy);


}
void cgbmv(char transA, int m, int n, int kl, int ku, float_complex alpha, float_complex* A, int lda, float_complex* X, int incx, float_complex beta, float_complex* Y, int incy) 
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasCgbmv(handle, convert_to_cublas_trans(transA), m, n, kl, ku, &alpha, A, lda, X, incx, &beta, Y, incy);


}
void zgbmv(char transA, int m, int n, int kl, int ku, double_complex alpha, double_complex* A, int lda, double_complex* X, int incx, double_complex beta, double_complex* Y, int incy) 
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasZgbmv(handle, convert_to_cublas_trans(transA), m, n, kl, ku, &alpha, A, lda, X, incx, &beta, Y, incy);


}




//XSYMV

void ssymv(char UPLO, int n, float alpha, float* A, int lda, float* X, int incx, float beta, float* Y, int incy)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasSsymv(handle, convert_fill(UPLO), n, &alpha, A, lda, X, incx, &beta, Y, incy);


}
void dsymv(char UPLO, int n, double alpha, double* A, int lda, double* X, int incx, double beta, double* Y, int incy)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasDsymv(handle, convert_fill(UPLO), n, &alpha, A, lda, X, incx, &beta, Y, incy);


}

void csymv(char UPLO, int n, float_complex alpha, float_complex* A, int lda, float_complex* X, int incx, float_complex beta, float_complex* Y, int incy)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasCsymv(handle, convert_fill(UPLO), n, &alpha, A, lda, X, incx, &beta, Y, incy);


}
void zsymv(char UPLO, int n, double_complex alpha, double_complex* A, int lda, double_complex* X, int incx, double_complex beta, double_complex* Y, int incy)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasZsymv(handle, convert_fill(UPLO), n, &alpha, A, lda, X, incx, &beta, Y, incy);


}



//XSPMV

void sspmv(char UPLO, int n, float alpha, float* AP, float* X, int incx, float beta, float* Y, int inc) 
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasSspmv(handle, convert_fill(UPLO), n, &alpha, AP, X, incx, &beta, Y, inc);


}
void dspmv(char UPLO, int n, double alpha, double* AP, double* X, int incx, double beta, double* Y, int inc) 
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasDspmv(handle, convert_fill(UPLO), n, &alpha, AP, X, incx, &beta, Y, inc);


}







//XTRMV

void strmv(char UPLO, char transA, char DIAG, int n, float* A, int lda, float* X, int incx)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasStrmv(handle, convert_fill(UPLO), convert_to_cublas_trans(transA), convert_diag(DIAG), n, A, lda, X, incx);


}
void dtrmv(char UPLO, char transA, char DIAG, int n, double* A, int lda, double* X, int incx)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasDtrmv(handle, convert_fill(UPLO), convert_to_cublas_trans(transA), convert_diag(DIAG), n, A, lda, X, incx);


}
void ctrmv(char UPLO, char transA, char DIAG, int n, float_complex* A, int lda, float_complex* X, int incx)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasCtrmv(handle, convert_fill(UPLO), convert_to_cublas_trans(transA), convert_diag(DIAG), n, A, lda, X, incx);


}
void ztrmv(char UPLO, char transA, char DIAG, int n, double_complex* A, int lda, double_complex* X, int incx)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasZtrmv(handle, convert_fill(UPLO), convert_to_cublas_trans(transA), convert_diag(DIAG), n, A, lda, X, incx);


}




//XTBMV

void stbmv(char UPLO, char TransA, char DIAG, int n, int k, float* A, int lda, float* X, int incx)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasStbmv(handle, convert_fill(UPLO), convert_to_cublas_trans(transA), convert_diag(DIAG), n, k, A, lda, X, incx);


}


void dtbmv(char UPLO, char TransA, char DIAG, int n, int k, double* A, int lda, double* X, int incx)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasDtbmv(handle, convert_fill(UPLO), convert_to_cublas_trans(transA), convert_diag(DIAG), n, k, A, lda, X, incx);


}
void ctbmv(char UPLO, char TransA, char DIAG, int n, int k, float_complex* A, int lda, float_complex* X, int incx)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasCtbmv(handle, convert_fill(UPLO), convert_to_cublas_trans(transA), convert_diag(DIAG), n, k, A, lda, X, incx);


}
void ztbmv(char UPLO, char TransA, char DIAG, int n, int k, double_complex* A, int lda, double_complex* X, int incx)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasZtbmv(handle, convert_fill(UPLO), convert_to_cublas_trans(transA), convert_diag(DIAG), n, k, A, lda, X, incx);


}




//XTPMV

void stpmv(char UPLO, char transA, char DIAG, int n, float* AP, float* X, int incx)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasStpmv(handle, convert_fill(UPLO), convert_to_cublas_trans(transA), convert_diag(DIAG), n, AP, lda, X, incx);


}
void dtpmv(char UPLO, char transA, char DIAG, int n, double* AP, double* X, int incx)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasDtpmv(handle, convert_fill(UPLO), convert_to_cublas_trans(transA), convert_diag(DIAG), n, AP, lda, X, incx);


}
void ctpmv(char UPLO, char transA, char DIAG, int n, float_complex* AP, float_complex* X, int incx)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasCtpmv(handle, convert_fill(UPLO), convert_to_cublas_trans(transA), convert_diag(DIAG), n, AP, lda, X, incx);


}
void ztpmv(char UPLO, char transA, char DIAG, int n, double_complex* AP, double_complex* X, int incx)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasZtpmv(handle, convert_fill(UPLO), convert_to_cublas_trans(transA), convert_diag(DIAG), n, AP, lda, X, incx);


}





/******
  BLAS 3
  *****/


//Xgemm


void dgemm(char transA, char transB, int M, int N, int K, double ALPHA, double* A, int LDA, double* B, int LDB, double BETA, double* C, int64_t LDC)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasDgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);


}	



void sgemm(char transA, char transB, int M, int N, int K, const float ALPHA, float* A, int LDA, float* B, int LDB, const float BETA, float* C, int LDC)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasSgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);	

		
}


void hgemm(char transA, char transB, int M, int N, int K, half ALPHA, half* A, int LDA, half* B, int LDB, half BETA, half* C, int LDC)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasHgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);	
	
}



void cgemm(char transA, char transB, int M, int N, int K, float_complex ALPHA, float_complex* A, int LDA, float_complex* B, int LDB, float_complex BETA, float_complex* C, int LDC)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasCgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);	

	
}


void zgemm(char transA, char transB, int M, int N, int K, double_complex ALPHA, double_complex* A, int LDA, double_complex* B, int LDB, double_complex BETA, double_complex* C, int LDC)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasZgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);	
		
}



void ssymm(char SIDE, char UPLO, int m, int n, float alpha, float* A, int lda, float* B, int ldb, float beta, float* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuSsymm(handle, convert_side(SIDE), convert_FILL(UPLO), m, n, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}
void dsymm(char SIDE, char UPLO, int m, int n, double alpha, double* A, int lda, double* B, int ldb, double beta, double* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuDsymm(handle, convert_side(SIDE), convert_FILL(UPLO), m, n, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}
void csymm(char SIDE, char UPLO, int m, int n, float_complex alpha, float_complex* A, int lda, float_complex* B, int ldb, float_complex beta, float_complex* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuCsymm(handle, convert_side(SIDE), convert_FILL(UPLO), m, n, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}
void zsymm(char SIDE, char UPLO, int m, int n, double_complex alpha, double_complex* A, int lda, double_complex* B, int ldb, double_complex beta, double_complex* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsymm(handle, convert_side(SIDE), convert_FILL(UPLO), m, n, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}



//Xsyrk

void ssyrk(char UPLO, char transA, int n, int k, float alpha, float* A, int lda, float beta, float* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsymm(handle, convert_FILL(UPLO), convert_to_cublas_trans(transA), n, k, &alpha, A, lda,&beta, C, ldc);	
		
}


void dsyrk(char UPLO, char transA, int n, int k, double alpha, double* A, int lda, double beta, double* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsymm(handle, convert_FILL(UPLO), convert_to_cublas_trans(transA), n, k, &alpha, A, lda,&beta, C, ldc);	
		
}
void csyrk(char UPLO, char transA, int n, int k, float_complex alpha, float_complex* A, int lda, float_complex beta, float_complex* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsymm(handle, convert_FILL(UPLO), convert_to_cublas_trans(transA), n, k, &alpha, A, lda,&beta, C, ldc);	
		
}
void zsyrk(char UPLO, char transA, int n, int k, double_complex alpha, double_complex* A, int lda, double_complex beta, double_complex* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsymm(handle, convert_FILL(UPLO), convert_to_cublas_trans(transA), n, k, &alpha, A, lda,&beta, C, ldc);	
		
}




//Xsyr2k



void ssyr2k(char UPLO, char TransA, int n, int k, float alpha, float* A, int lda, float* B, int ldb, float beta, float* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsyr2k(handle, convert_FILL(UPLO), convert_to_cublas_trans(transA), n, k, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}


void dsyr2k(char UPLO, char TransA, int n, int k, double alpha, double* A, int lda, double* B, int ldb, double beta, double* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsyr2k(handle, convert_FILL(UPLO), convert_to_cublas_trans(transA), n, k, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}
void csyr2k(char UPLO, char TransA, int n, int k, float_complex alpha, float_complex* A, int lda, float_complex* B, int ldb, float_complex beta, float_complex* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsyr2k(handle, convert_FILL(UPLO), convert_to_cublas_trans(transA), n, k, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}
void zsyr2k(char UPLO, char TransA, int n, int k, double_complex alpha, double_complex* A, int lda, double_complex* B, int ldb, double_complex beta, double_complex* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsyr2k(handle, convert_FILL(UPLO), convert_to_cublas_trans(transA), n, k, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}

//Xtrmm



void strmm(char SIDE, char UPLO, char TransA, char DIAG, int m, int n, float alpha, float* A, int lda, float* B, int ldb);
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasStrmm(handle, convert_side(SIDE), convert_fill(UPLO), convert_to_cublas_trans(TransA), convert_diag(DIAG), m, n, &alpha, A, lda, B, ldb);
}
void dtrmm(char SIDE, char UPLO, char TransA, char DIAG, int m, int n, double alpha, double* A, int lda, double* B, int ldb);
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasDtrmm(handle, convert_side(SIDE), convert_fill(UPLO), convert_to_cublas_trans(TransA), convert_diag(DIAG), m, n, &alpha, A, lda, B, ldb);
}
void ctrmm(char SIDE, char UPLO, char TransA, char DIAG, int m, int n, float_complex alpha, float_complex* A, int lda, float_complex* B, int ldb);
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasCtrmm(handle, convert_side(SIDE), convert_fill(UPLO), convert_to_cublas_trans(TransA), convert_diag(DIAG), m, n, &alpha, A, lda, B, ldb);
}
void ztrmm(char SIDE, char UPLO, char TransA, char DIAG, int m, int n, double_complex alpha, double_complex* A, int lda, double_complex* B, int ldb);
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasZtrmm(handle, convert_side(SIDE), convert_fill(UPLO), convert_to_cublas_trans(TransA), convert_diag(DIAG), m, n, &alpha, A, lda, B, ldb);
}





//Xtrsm


void strsm(char SIDE, char UPLO, char TransA, char DIAG, int  m, int  n, float alpha, float* A, int LDA, float* B, int ldb);
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasStrsm(handle, convert_side(SIDE), convert_fill(UPLO), convert_to_cublas_trans(TransA), convert_diag(DIAG), m, n, &alpha, A, lda, B, ldb);
}


void dtrsm(char SIDE, char UPLO, char TransA, char DIAG, int  m, int  n, double alpha, double* A, int LDA, double* B, int ldb);
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasDtrsm(handle, convert_side(SIDE), convert_fill(UPLO), convert_to_cublas_trans(TransA), convert_diag(DIAG), m, n, &alpha, A, lda, B, ldb);
}


void ctrsm(char SIDE, char UPLO, char TransA, char DIAG, int  m, int  n, float_complex alpha, float_complex* A, int LDA, float_complex* B, int ldb);
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasCtrsm(handle, convert_side(SIDE), convert_fill(UPLO), convert_to_cublas_trans(TransA), convert_diag(DIAG), m, n, &alpha, A, lda, B, ldb);
}


void ztrsm(char SIDE, char UPLO, char TransA, char DIAG, int  m, int  n, double_complex alpha, double_complex* A, int LDA, double_complex* B, int ldb) ;
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasZtrsm(handle, convert_side(SIDE), convert_fill(UPLO), convert_to_cublas_trans(TransA), convert_diag(DIAG), m, n, &alpha, A, lda, B, ldb);
}




