#include "../include/cublas.hpp"

void cublas_manage_error(int error_number)
{
	if(error_number == 1)
	{
		printf("Bad transformation type");
	}
}

hipblasHandle_t init_cublas()
{

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	return handle;

}

hipblasOperation_t convert_to_cublas_trans(char trans)
{
	hipblasOperation_t ret;
	if(trans == 'N' || trans == 'n'){
		 ret = HIPBLAS_OP_N; 
	}else if(trans == 'C' || trans == 'c')
	{
		ret = HIPBLAS_OP_C;
	}else if(trans == 'T' || trans == 't')
	{
		ret = HIPBLAS_OP_T;
	}else
	{
		cublas_manage_error(1);
	}	
	
	return ret;
}

void dgemm(char transA, char transB, int M, int N, int K, double ALPHA, double* A, int LDA, double* B, int LDB, double BETA, double* C, int64_t LDC)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasDgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);


}	



void sgemm(char transA, char transB, int M, int N, int K, const float ALPHA, float* A, int LDA, float* B, int LDB, const float BETA, float* C, int LDC)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasSgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);	

		
}


void hgemm(char transA, char transB, int M, int N, int K, half ALPHA, half* A, int LDA, half* B, int LDB, half BETA, half* C, int LDC)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasHgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);	
	
}



void cgemm(char transA, char transB, int M, int N, int K, float_complex ALPHA, float_complex* A, int LDA, float_complex* B, int LDB, float_complex BETA, float_complex* C, int LDC)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret =rocblas_cgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);	

	
}


void zgemm(char transA, char transB, int M, int N, int K, double_complex ALPHA, double_complex* A, int LDA, double_complex* B, int LDB, double_complex BETA, double_complex* C, int LDC)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret =rocblas_zgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);	
		
}
