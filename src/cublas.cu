#include "../include/cublas.hpp"

void cublas_manage_error(int error_number)
{
	if(error_number == 1)
	{
		printf("Bad transformation type");
	}
}

hipblasHandle_t init_cublas()
{

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	return handle;

}

hipblasOperation_t convert_to_cublas_trans(char trans)
{
	hipblasOperation_t ret;
	if(trans == 'N' || trans == 'n'){
		 ret = HIPBLAS_OP_N; 
	}else if(trans == 'C' || trans == 'c')
	{
		ret = HIPBLAS_OP_C;
	}else if(trans == 'T' || trans == 't')
	{
		ret = HIPBLAS_OP_T;
	}else
	{
		cublas_manage_error(1);
	}	
	
	return ret;
}


cublasSideMode convert_side(char* SIDE)
{
	cublasSideMode ret;
	if(SIDE == 'L' || side == 'l')
		ret = HIPBLAS_SIDE_LEFT;
	else if(SIDE == 'r' || SIDE == 'R')
		ret = HIPBLAS_SIDE_RIGHT;
	return ret
}
cublasFillMode convert_fill(char* UPLO)
{
	cublasFillMode ret;
	if(UPLO == 'U' || UPLO == 'u')
		ret == HIPBLAS_FILL_MODE_UPPER;
	else if(UPLO == 'L' || UPLO == 'l')
		ret == HIPBLAS_FILL_MODE_LOWER;
	return ret;

}



void dgemm(char transA, char transB, int M, int N, int K, double ALPHA, double* A, int LDA, double* B, int LDB, double BETA, double* C, int64_t LDC)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasDgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);


}	



void sgemm(char transA, char transB, int M, int N, int K, const float ALPHA, float* A, int LDA, float* B, int LDB, const float BETA, float* C, int LDC)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasSgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);	

		
}


void hgemm(char transA, char transB, int M, int N, int K, half ALPHA, half* A, int LDA, half* B, int LDB, half BETA, half* C, int LDC)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasHgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);	
	
}



void cgemm(char transA, char transB, int M, int N, int K, float_complex ALPHA, float_complex* A, int LDA, float_complex* B, int LDB, float_complex BETA, float_complex* C, int LDC)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasCgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);	

	
}


void zgemm(char transA, char transB, int M, int N, int K, double_complex ALPHA, double_complex* A, int LDA, double_complex* B, int LDB, double_complex BETA, double_complex* C, int LDC)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasZgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);	
		
}



void ssymm(char SIDE, char UPLO, int m, int n, float alpha, float* A, int lda, float* B, int ldb, float beta, float* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuSsymm(handle, convert_side(SIDE), convert_FILL(UPLO), m, n, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}
void dsymm(char SIDE, char UPLO, int m, int n, double alpha, double* A, int lda, double* B, int ldb, double beta, double* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuDsymm(handle, convert_side(SIDE), convert_FILL(UPLO), m, n, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}
void csymm(char SIDE, char UPLO, int m, int n, float_complex alpha, float_complex* A, int lda, float_complex* B, int ldb, float_complex beta, float_complex* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuCsymm(handle, convert_side(SIDE), convert_FILL(UPLO), m, n, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}
void zsymm(char SIDE, char UPLO, int m, int n, double_complex alpha, double_complex* A, int lda, double_complex* B, int ldb, double_complex beta, double_complex* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsymm(handle, convert_side(SIDE), convert_FILL(UPLO), m, n, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}

