#include "../include/cublas.hpp"

void cublas_manage_error(int error_number)
{
	if(error_number == 1)
	{
		printf("Bad transformation type");
	}
}

hipblasHandle_t init_cublas()
{

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	return handle;

}

hipblasOperation_t convert_to_cublas_trans(char trans)
{
	hipblasOperation_t ret;
	if(trans == 'N' || trans == 'n'){
		 ret = HIPBLAS_OP_N; 
	}else if(trans == 'C' || trans == 'c')
	{
		ret = HIPBLAS_OP_C;
	}else if(trans == 'T' || trans == 't')
	{
		ret = HIPBLAS_OP_T;
	}else
	{
		cublas_manage_error(1);
	}	
	
	return ret;
}



hipblasDiagType_t convert_diag(char* DIAG)
{
	hipblasDiagType_t ret;
	if(DIAG == 'N' || DIAG == 'n')
		ret = HIPBLAS_DIAG_NON_UNIT;
	else if(DIAG == 'U' || DIAG == 'u')
		ret = HIPBLAS_DIAG_UNIT;

	return ret;
}

hipblasSideMode_t convert_side(char* SIDE)
{
	hipblasSideMode_t ret;
	if(SIDE == 'L' || side == 'l')
		ret = HIPBLAS_SIDE_LEFT;
	else if(SIDE == 'r' || SIDE == 'R')
		ret = HIPBLAS_SIDE_RIGHT;
	return ret
}
hipblasFillMode_t convert_fill(char* UPLO)
{
	hipblasFillMode_t ret;
	if(UPLO == 'U' || UPLO == 'u')
		ret == HIPBLAS_FILL_MODE_UPPER;
	else if(UPLO == 'L' || UPLO == 'l')
		ret == HIPBLAS_FILL_MODE_LOWER;
	return ret;

}




//XGEMV


void sgemv(char transA, int m, int n, float alpha, float* A, int lda, float* X, int incx, float beta, float* Y, int incy); 
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasSgemv(handle, convert_to_cublas_trans(transA), m, n, &alpha, A, lda, X, incx, &beta, Y, incy);


}
void dgemv(char transA, int m, int n, double alpha, double* A, int lda, double* X, int incx, double beta, double* Y, int incy) ;
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasDgemv(handle, convert_to_cublas_trans(transA), m, n, &alpha, A, lda, X, incx, &beta, Y, incy);


}

void cgemv(char transA, int m, int n, float_complex alpha, float_complex* A, int lda, float_complex* X, int incx, float_complex beta, float_complex* Y, int incy);
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasCgemv(handle, convert_to_cublas_trans(transA), m, n, &alpha, A, lda, X, incx, &beta, Y, incy);


}

void zgemv(char transA, int m, int n, double_complex alpha, double_complex* A, int lda, double_complex* X, int incx, double_complex beta, double_complex* Y, int incy);
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasZgemv(handle, convert_to_cublas_trans(transA), m, n, &alpha, A, lda, X, incx, &beta, Y, incy);


}



//XGBMV

void sgbmv(char transA, int m, int n, int kl, int ku, float alpha, float* A, int lda, float* X, int incx, float beta, float* Y, int incy)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasZgemv(handle, convert_to_cublas_trans(transA), m, n, kl, ku, &alpha, A, lda, X, incx, &beta, Y, incy);


}

void dgbmv(char transA, int m, int n, int kl, int ku, double alpha, double* A, int lda, double* X, int incx, double beta, double* Y, int incy) 
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasZgemv(handle, convert_to_cublas_trans(transA), m, n, kl, ku, &alpha, A, lda, X, incx, &beta, Y, incy);


}
void cgbmv(char transA, int m, int n, int kl, int ku, float_complex alpha, float_complex* A, int lda, float_complex* X, int incx, float_complex beta, float_complex* Y, int incy) 
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasZgemv(handle, convert_to_cublas_trans(transA), m, n, kl, ku, &alpha, A, lda, X, incx, &beta, Y, incy);


}
void zgbmv(char transA, int m, int n, int kl, int ku, double_complex alpha, double_complex* A, int lda, double_complex* X, int incx, double_complex beta, double_complex* Y, int incy) 
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasZgemv(handle, convert_to_cublas_trans(transA), m, n, kl, ku, &alpha, A, lda, X, incx, &beta, Y, incy);


}





//Xgemm


void dgemm(char transA, char transB, int M, int N, int K, double ALPHA, double* A, int LDA, double* B, int LDB, double BETA, double* C, int64_t LDC)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasDgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);


}	



void sgemm(char transA, char transB, int M, int N, int K, const float ALPHA, float* A, int LDA, float* B, int LDB, const float BETA, float* C, int LDC)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasSgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);	

		
}


void hgemm(char transA, char transB, int M, int N, int K, half ALPHA, half* A, int LDA, half* B, int LDB, half BETA, half* C, int LDC)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasHgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);	
	
}



void cgemm(char transA, char transB, int M, int N, int K, float_complex ALPHA, float_complex* A, int LDA, float_complex* B, int LDB, float_complex BETA, float_complex* C, int LDC)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasCgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);	

	
}


void zgemm(char transA, char transB, int M, int N, int K, double_complex ALPHA, double_complex* A, int LDA, double_complex* B, int LDB, double_complex BETA, double_complex* C, int LDC)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasZgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);	
		
}



void ssymm(char SIDE, char UPLO, int m, int n, float alpha, float* A, int lda, float* B, int ldb, float beta, float* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuSsymm(handle, convert_side(SIDE), convert_FILL(UPLO), m, n, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}
void dsymm(char SIDE, char UPLO, int m, int n, double alpha, double* A, int lda, double* B, int ldb, double beta, double* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuDsymm(handle, convert_side(SIDE), convert_FILL(UPLO), m, n, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}
void csymm(char SIDE, char UPLO, int m, int n, float_complex alpha, float_complex* A, int lda, float_complex* B, int ldb, float_complex beta, float_complex* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuCsymm(handle, convert_side(SIDE), convert_FILL(UPLO), m, n, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}
void zsymm(char SIDE, char UPLO, int m, int n, double_complex alpha, double_complex* A, int lda, double_complex* B, int ldb, double_complex beta, double_complex* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsymm(handle, convert_side(SIDE), convert_FILL(UPLO), m, n, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}



//Xsyrk

void ssyrk(char UPLO, char transA, int n, int k, float alpha, float* A, int lda, float beta, float* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsymm(handle, convert_FILL(UPLO), convert_to_cublas_trans(transA), n, k, &alpha, A, lda,&beta, C, ldc);	
		
}


void dsyrk(char UPLO, char transA, int n, int k, double alpha, double* A, int lda, double beta, double* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsymm(handle, convert_FILL(UPLO), convert_to_cublas_trans(transA), n, k, &alpha, A, lda,&beta, C, ldc);	
		
}
void csyrk(char UPLO, char transA, int n, int k, float_complex alpha, float_complex* A, int lda, float_complex beta, float_complex* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsymm(handle, convert_FILL(UPLO), convert_to_cublas_trans(transA), n, k, &alpha, A, lda,&beta, C, ldc);	
		
}
void zsyrk(char UPLO, char transA, int n, int k, double_complex alpha, double_complex* A, int lda, double_complex beta, double_complex* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsymm(handle, convert_FILL(UPLO), convert_to_cublas_trans(transA), n, k, &alpha, A, lda,&beta, C, ldc);	
		
}




//Xsyr2k



void ssyr2k(char UPLO, char TransA, int n, int k, float alpha, float* A, int lda, float* B, int ldb, float beta, float* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsyr2k(handle, convert_FILL(UPLO), convert_to_cublas_trans(transA), n, k, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}


void dsyr2k(char UPLO, char TransA, int n, int k, double alpha, double* A, int lda, double* B, int ldb, double beta, double* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsyr2k(handle, convert_FILL(UPLO), convert_to_cublas_trans(transA), n, k, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}
void csyr2k(char UPLO, char TransA, int n, int k, float_complex alpha, float_complex* A, int lda, float_complex* B, int ldb, float_complex beta, float_complex* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsyr2k(handle, convert_FILL(UPLO), convert_to_cublas_trans(transA), n, k, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}
void zsyr2k(char UPLO, char TransA, int n, int k, double_complex alpha, double_complex* A, int lda, double_complex* B, int ldb, double_complex beta, double_complex* C, int ldc)
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = cuZsyr2k(handle, convert_FILL(UPLO), convert_to_cublas_trans(transA), n, k, &alpha, A, lda, B, ldb, &beta, C, ldc);	
		
}

//Xtrmm



void strmm(char SIDE, char UPLO, char TransA, char DIAG, int m, int n, float alpha, float* A, int lda, float* B, int ldb);
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasStrmm(handle, convert_side(SIDE), convert_fill(UPLO), convert_to_cublas_trans(TransA), convert_diag(DIAG), m, n, &alpha, A, lda, B, ldb);
}
void dtrmm(char SIDE, char UPLO, char TransA, char DIAG, int m, int n, double alpha, double* A, int lda, double* B, int ldb);
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasDtrmm(handle, convert_side(SIDE), convert_fill(UPLO), convert_to_cublas_trans(TransA), convert_diag(DIAG), m, n, &alpha, A, lda, B, ldb);
}
void ctrmm(char SIDE, char UPLO, char TransA, char DIAG, int m, int n, float_complex alpha, float_complex* A, int lda, float_complex* B, int ldb);
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasCtrmm(handle, convert_side(SIDE), convert_fill(UPLO), convert_to_cublas_trans(TransA), convert_diag(DIAG), m, n, &alpha, A, lda, B, ldb);
}
void ztrmm(char SIDE, char UPLO, char TransA, char DIAG, int m, int n, double_complex alpha, double_complex* A, int lda, double_complex* B, int ldb);
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasZtrmm(handle, convert_side(SIDE), convert_fill(UPLO), convert_to_cublas_trans(TransA), convert_diag(DIAG), m, n, &alpha, A, lda, B, ldb);
}





//Xtrsm


void strsm(char SIDE, char UPLO, char TransA, char DIAG, int  m, int  n, float alpha, float* A, int LDA, float* B, int ldb);
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasStrsm(handle, convert_side(SIDE), convert_fill(UPLO), convert_to_cublas_trans(TransA), convert_diag(DIAG), m, n, &alpha, A, lda, B, ldb);
}


void dtrsm(char SIDE, char UPLO, char TransA, char DIAG, int  m, int  n, double alpha, double* A, int LDA, double* B, int ldb);
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasDtrsm(handle, convert_side(SIDE), convert_fill(UPLO), convert_to_cublas_trans(TransA), convert_diag(DIAG), m, n, &alpha, A, lda, B, ldb);
}


void ctrsm(char SIDE, char UPLO, char TransA, char DIAG, int  m, int  n, float_complex alpha, float_complex* A, int LDA, float_complex* B, int ldb);
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasCtrsm(handle, convert_side(SIDE), convert_fill(UPLO), convert_to_cublas_trans(TransA), convert_diag(DIAG), m, n, &alpha, A, lda, B, ldb);
}


void ztrsm(char SIDE, char UPLO, char TransA, char DIAG, int  m, int  n, double_complex alpha, double_complex* A, int LDA, double_complex* B, int ldb) ;
{

	void* handle = (hipblasHandle_t)init_cublas();

	ret = hipblasZtrsm(handle, convert_side(SIDE), convert_fill(UPLO), convert_to_cublas_trans(TransA), convert_diag(DIAG), m, n, &alpha, A, lda, B, ldb);
}




