#include "../include/cublas.hpp"

void cublas_manage_error(int error_number)
{
	if(error_number == 1)
	{
		printf("Bad transformation type");
	}
}

hipblasHandle_t init_cublas()
{

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	return handle;

}

hipblasOperation_t convert_to_cublas_trans(char trans)
{
	hipblasOperation_t ret;
	if(trans == 'N' || trans == 'n'){
		 ret = HIPBLAS_OP_N; 
	}else if(trans == 'C' || trans == 'c')
	{
		ret = HIPBLAS_OP_C;
	}else if(trans == 'T' || trans == 't')
	{
		ret = HIPBLAS_OP_T;
	}else
	{
		cublas_manage_error(1);
	}	
	
	return ret;
}

void dgemm(char transA, char transB, int M, int N, int K, double ALPHA, double* A, int LDA, double* B, int LDB, double BETA, double* C, int64_t LDC)
{

	hipblasHandle_t handle = (hipblasHandle_t)init_cublas();
	
	hipblasStatus_t error = hipblasDgemm(handle, convert_to_cublas_trans(transA), convert_to_cublas_trans(transB), M, N, K, &ALPHA, A, LDA, B, LDB, &BETA, C, LDC);



	

}	
